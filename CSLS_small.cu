#include "hip/hip_runtime.h"
#include "CSLS_small.cuh"

CUDA_SLS::CUDA_SLS(const std::string& file_name, const bool read_b, const bool sym_triang) : mat(file_name, read_b, sym_triang){
  //Constructor that constructs a SparseMatrix SM
}

void CUDA_SLS::mult(double* result, double* vector){
    if(mult_algo == "staircase"){
    //Launch kernels with 1024 threads in each to achieve maximal coalescence during read and write
    for (int i = 0; i < num_steps; ++i) {
        staircase_mult<<<step_blocks[i], slice_height, 0, streams[i]>>>(step_lengths[i], row_starts[i], step_starts[i], n_rows);
    }
    //hipDeviceSynchronize();
    CUDA_CHECK
  }

  //Sliced Ellpack format
  else if(mult_algo == "ellpack_shared"){
    ellpack_shared<<<blocks, tile, tile.x*sizeof(double)>>>(diag_dev, entries_dev, col_dev, slice_ptr_dev, vector, result, n_rows);
    //CUDA_CHECK
  }

  else if(mult_algo == "ellpack_row_based"){
    ellpack_row_based<<<blocks, tile, tile.x*sizeof(double)>>>(entries_dev, col_dev, slice_ptr_dev, vector, result, n_rows);
    //CUDA_CHECK
  }
}

//Copy system of equations, i.e. matrix and RHS vector to GPU
//Get also the matrix storage format/multiplication algorithm
void CUDA_SLS::copy_system(std::string algo, double* b){
  n_rows = mat.n_rows;
  slice_height =  mat.slice_height;

  hipMalloc((void**)&b_dev, sizeof(double) * n_rows);
  CUDA_CHECK
  hipMemcpy(b_dev, b, sizeof(double) * n_rows, hipMemcpyHostToDevice);
  CUDA_CHECK

  vector_blocks.x = (n_rows + 1023)/1024;
  hipMalloc((void**)&reduc_arr1, vector_blocks.x*sizeof(double));
  CUDA_CHECK
  hipMalloc((void**)&reduc_arr2, ((vector_blocks.x + 1023)/1024)*sizeof(double));
  CUDA_CHECK

  if(algo == "staircase"){
    num_steps = mat.nr_steps;
    streams = new hipStream_t[num_steps];
    for (int i = 0; i < num_steps; ++i) {
        hipStreamCreate(&streams[i]);
    }

    mult_algo = "staircase";

    step_lengths = new int[num_steps];
    row_starts = new int[num_steps + 1]();
    step_starts = new int[num_steps + 1]();
    step_blocks = new int[num_steps];

    for(int i = 0; i < num_steps + 1; i++){
      if(i < num_steps){
        step_lengths[i] = mat.step_lengths[i];
        step_blocks[i] = mat.step_blocks[i];
      }

      row_starts[i] = mat.row_starts[i];
      step_starts[i] = mat.step_starts[i];
    }

    //allocate space
    hipMalloc((void**)&entries_dev, sizeof(double) * mat.entries_size);
    CUDA_CHECK
    hipMalloc((void**)&col_dev, sizeof(int) * mat.entries_size);
    CUDA_CHECK

    //Copy onto device
    hipMemcpy(entries_dev, mat.entries, sizeof(double) * mat.entries_size,
           hipMemcpyHostToDevice);
    CUDA_CHECK
    hipMemcpy(col_dev, mat.col_idx, sizeof(int) * mat.entries_size,
           hipMemcpyHostToDevice);
    CUDA_CHECK

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(entries_ptr), &entries_dev, sizeof(double*)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(col_ptr), &col_dev, sizeof(int*)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(x_ptr), &d_dev, sizeof(double*)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(y_ptr), &Ad_dev, sizeof(double*)));
  }

  else if(algo == "ellpack_shared"){
    tile.x = slice_height;
    blocks.x = (n_rows + slice_height - 1)/slice_height;

    mult_algo = "ellpack_shared";

    smooth_smem_size = slice_height*(sizeof(double) + mat.max_slice_mnz*(sizeof(int) + sizeof(double)));

    //allocate space
    hipMalloc((void**)&entries_dev, sizeof(double) * mat.entries_size);
    CUDA_CHECK
    hipMalloc((void**)&diag_dev, sizeof(double) * n_rows);
    CUDA_CHECK
    hipMalloc((void**)&col_dev, sizeof(int) * mat.entries_size);
    CUDA_CHECK
    hipMalloc((void**)&slice_ptr_dev, sizeof(int) * (mat.num_slices + 1));
    CUDA_CHECK
    hipMalloc((void**)&slice_mnz_dev, sizeof(int) * mat.num_slices);
    CUDA_CHECK

    //Copy onto device
    hipMemcpy(entries_dev, mat.entries, sizeof(double) * mat.entries_size,
           hipMemcpyHostToDevice);
    CUDA_CHECK
    hipMemcpy(diag_dev, mat.diag, sizeof(double) * mat.n_rows,
           hipMemcpyHostToDevice);
    CUDA_CHECK
    hipMemcpy(col_dev, mat.col_idx, sizeof(int) * mat.entries_size,
           hipMemcpyHostToDevice);
    CUDA_CHECK
    hipMemcpy(slice_ptr_dev, mat.sellp_slice_ptr, sizeof(int) * (mat.num_slices + 1),
           hipMemcpyHostToDevice);
    CUDA_CHECK

    hipMemcpy(slice_mnz_dev, mat.slice_mnz, sizeof(int) * mat.num_slices,
           hipMemcpyHostToDevice);
    CUDA_CHECK
  }

  else if(algo == "ellpack_row_based"){
    tile.x = slice_height;
    blocks.x = (n_rows + slice_height - 1)/slice_height;

    mult_algo = "ellpack_row_based";

    //allocate space
    hipMalloc((void**)&entries_dev, sizeof(double) * mat.entries_size);
    CUDA_CHECK
    hipMalloc((void**)&col_dev, sizeof(int) * mat.entries_size);
    CUDA_CHECK
    hipMalloc((void**)&slice_ptr_dev, sizeof(int) * (mat.num_slices + 1));
    CUDA_CHECK

    //Copy onto device
    hipMemcpy(entries_dev, mat.entries, sizeof(double) * mat.entries_size,
           hipMemcpyHostToDevice);
    CUDA_CHECK
    hipMemcpy(col_dev, mat.col_idx, sizeof(int) * mat.entries_size,
           hipMemcpyHostToDevice);
    CUDA_CHECK
    hipMemcpy(slice_ptr_dev, mat.sellp_slice_ptr, sizeof(int) * (mat.num_slices + 1),
           hipMemcpyHostToDevice);
    CUDA_CHECK
  }
}

double CUDA_SLS::calc_res(){
  mult(Ad_dev, x_dev);
  add_kern<<<vector_blocks.x, 1024>>>(Ad_dev, b_dev, Ad_dev, n_rows, true);
  //CUDA_CHECK

  dot_prod(rTr_dev, Ad_dev);
  //CUDA_CHECK

  double* rTr_h = new double[1];

  hipMemcpy(rTr_h, rTr_dev, sizeof(double),hipMemcpyDeviceToHost);
  //CUDA_CHECK

  double rTr = rTr_h[0];
  delete[] rTr_h;

  return rTr;
}

void CUDA_SLS::prepare_smoother(bool async){
  if(x_dev == nullptr){
    hipMalloc((void**)&x_dev, n_rows*sizeof(double));
    CUDA_CHECK
  }

  if(Ad_dev == nullptr){
    hipMalloc((void**)&Ad_dev, sizeof(double)*n_rows);
    CUDA_CHECK
  }

  if(rTr_dev == nullptr){
    hipMalloc((void**)&rTr_dev, sizeof(double));
    CUDA_CHECK
  }

  if(!async && x_dev_new == nullptr){
    hipMalloc((void**)&x_dev_new, n_rows*sizeof(double));
    CUDA_CHECK
  }

  dot_prod(rTr_dev, b_dev);
  CUDA_CHECK
}

void CUDA_SLS::reset_smoother(){
  hipMemset(x_dev, 0.0, sizeof(double)*n_rows);
}

//Make a Conjugate-Gradient Step on GPU
void CUDA_SLS::CG_step(){
  mult(Ad_dev, d_dev);
  //CUDA_CHECK

  dot_prod(dAd_dev, d_dev, Ad_dev);
  //CUDA_CHECK

  add_kern<<<blocks, tile>>>(x_dev, x_dev, d_dev, n_rows, false, rTr_dev, dAd_dev);
  //CUDA_CHECK

  add_kern<<<blocks, tile>>>(res_dev, res_dev, Ad_dev, n_rows, true, rTr_dev, dAd_dev);
  //CUDA_CHECK

  dot_prod(nrTnr_dev, res_dev);
  //CUDA_CHECK
  add_kern<<<blocks, tile>>>(d_dev, res_dev, d_dev, n_rows, false, nrTnr_dev, rTr_dev);
  //CUDA_CHECK

  std::swap(nrTnr_dev, rTr_dev);
}

//Recompute residual and set b and d to this residual. Set x to zero
void CUDA_SLS::reset_CG(){
    mult(Ad_dev, x_dev);
    CUDA_CHECK
    add_kern<<<vector_blocks, 1024>>>(res_dev, b_dev, Ad_dev, n_rows, true);
    CUDA_CHECK
    hipMemcpy(b_dev, res_dev, sizeof(double)*n_rows, hipMemcpyDeviceToDevice);
    CUDA_CHECK
    hipMemcpy(d_dev, res_dev, sizeof(double)*n_rows, hipMemcpyDeviceToDevice);
    CUDA_CHECK
    hipMemset(x_dev, 0.0, sizeof(double)*n_rows);
    CUDA_CHECK
    dot_prod(rTr_dev, res_dev);
    CUDA_CHECK
}

void CUDA_SLS::prepare_CG(double* x, bool x_not_precond){
  if(d_dev == nullptr){
    hipMalloc((void**)&d_dev, sizeof(double)*n_rows);
    CUDA_CHECK
  }

  if(res_dev == nullptr){
    hipMalloc((void**)&res_dev, sizeof(double)*n_rows);
    CUDA_CHECK
  }

  if(Ad_dev == nullptr){
    hipMalloc((void**)&Ad_dev, sizeof(double)*n_rows);
    CUDA_CHECK
  }

  if(dAd_dev == nullptr){
    hipMalloc((void**)&dAd_dev, sizeof(double));
    CUDA_CHECK
  }

  if(rTr_dev == nullptr){
    hipMalloc((void**)&rTr_dev, sizeof(double));
    CUDA_CHECK
  }

  if(nrTnr_dev == nullptr){
    hipMalloc((void**)&nrTnr_dev, sizeof(double));
    CUDA_CHECK
  }

  if(x_dev == nullptr){
    hipMalloc((void**)&x_dev, sizeof(double)*n_rows);
    CUDA_CHECK
  }

  //If x-guess, x' given, set b -= Ax'
  if(x!=nullptr){
    hipMemcpy(x_dev, x, sizeof(double)*n_rows, hipMemcpyHostToDevice);
    CUDA_CHECK
    if(x_not_precond && is_precond){
      D_scale_kern<<<vector_blocks, 1024>>>(D_dev, x_dev, n_rows);
      CUDA_CHECK
    }

    reset_CG();
    CUDA_CHECK
  }

  else {
    hipMemcpy(res_dev, b_dev, sizeof(double)*n_rows, hipMemcpyDeviceToDevice);
    CUDA_CHECK
    hipMemcpy(d_dev, b_dev, sizeof(double)*n_rows, hipMemcpyDeviceToDevice);
    CUDA_CHECK
    hipMemset(x_dev, 0.0, sizeof(double)*n_rows);
    CUDA_CHECK
    dot_prod(rTr_dev, res_dev);
    CUDA_CHECK
  }
}

void CUDA_SLS::prepare_diag_precond(){
  if(rTr_orig_dev == nullptr){
    hipMalloc((void**)&rTr_orig_dev, sizeof(double));
    CUDA_CHECK
    rTr_orig_h = new double[1];
  }

  if(D_dev == nullptr){
    hipMalloc((void**)&D_dev, sizeof(double)*n_rows);
    CUDA_CHECK
  }

  if(orig_res_dev == nullptr){
    hipMalloc((void**)&orig_res_dev, sizeof(double)*n_rows);
    CUDA_CHECK
  }
}

void CUDA_SLS::diag_precond(){
  is_precond = true;

  hipMemcpy(D_dev, diag_dev, n_rows*sizeof(double), hipMemcpyDeviceToDevice);
  //CUDA_CHECK

  Dinv_scale_kern<<<vector_blocks, 1024>>>(D_dev, b_dev, n_rows);
  //CUDA_CHECK
  scale_sell<<<blocks, tile, sizeof(double)*tile.x>>>(diag_dev, entries_dev, col_dev, slice_ptr_dev, D_dev, n_rows);
  //CUDA_CHECK
}

void CUDA_SLS::prepare_Ruiz_precond(){
  is_precond = true;

  if(D_dev == nullptr){
    hipMalloc((void**)&D_dev, sizeof(double)*n_rows);
    CUDA_CHECK
  }

  if(orig_res_dev == nullptr){
    hipMalloc((void**)&orig_res_dev, sizeof(double)*n_rows);
    CUDA_CHECK
  }

  double* D_h = new double[n_rows];
  std::fill(D_h, D_h + n_rows, 1.0);

  hipMemcpy(D_dev, D_h, n_rows*sizeof(double), hipMemcpyHostToDevice);
  CUDA_CHECK

  delete[] D_h;

  R_max_CPU = new double[1];

  if(R_max_dev == nullptr){
    hipMalloc((void**)&R_max_dev, sizeof(double));
    CUDA_CHECK
  }

  if(R_dev == nullptr){
    hipMalloc((void**)&R_dev, sizeof(double)*n_rows);
    CUDA_CHECK
  }

  if(rTr_orig_dev == nullptr){
    hipMalloc((void**)&rTr_orig_dev, sizeof(double));
    CUDA_CHECK
    rTr_orig_h = new double[1];
  }
}

int CUDA_SLS::precond_Ruiz(double equi_tol, double check_frequency){
  prepare_Ruiz_precond();
  double R_max_deviation = 2;
  int num_iters = 0;

  while(R_max_deviation > equi_tol){
    for(int i = 0; i < check_frequency; i++){
      scale_iter();
      //CUDA_CHECK
      num_iters +=1;
    }

    R_max_deviation = max_deviation();
  }

  Dinv_scale_kern<<<vector_blocks, 1024>>>(D_dev, b_dev, n_rows);

  //CUDA_CHECK
  return num_iters;
}

double CUDA_SLS::max_deviation(){
  max_deviation_kern<<<vector_blocks, 1024>>>(R_dev, n_rows, reduc_arr1);
  reduce(R_max_dev, true);
  hipMemcpy(R_max_CPU, R_max_dev, sizeof(double), hipMemcpyDeviceToHost);
  return R_max_CPU[0];
}

void CUDA_SLS::undo_precond(){
  if(is_precond){
    scale_inv_sell<<<blocks, tile, sizeof(double)*tile.x>>>(diag_dev, entries_dev, col_dev, slice_ptr_dev, D_dev, n_rows);
    D_scale_kern<<<vector_blocks, 1024>>>(D_dev, b_dev, n_rows);
  }
  is_precond = false;
}

CUDA_SLS::~CUDA_SLS(){
    if(entries_dev != nullptr){
      hipFree(entries_dev);
      CUDA_CHECK
    }

    if(col_dev != nullptr){
      hipFree(col_dev);
      CUDA_CHECK
    }

    if(res_dev != nullptr){
      hipFree(res_dev);
      CUDA_CHECK
    }

    if(d_dev != nullptr){
      hipFree(d_dev);
      CUDA_CHECK
    }

    if(Ad_dev != nullptr){
      hipFree(Ad_dev);
      CUDA_CHECK
    }

    if(dAd_dev != nullptr){
      hipFree(dAd_dev);
      CUDA_CHECK
    }

    if(rTr_dev != nullptr){
      hipFree(rTr_dev);
      CUDA_CHECK
    }

    if(nrTnr_dev != nullptr){
      hipFree(nrTnr_dev);
      CUDA_CHECK
    }

    if(x_dev != nullptr){
      hipFree(x_dev);
      CUDA_CHECK
    }

    if(slice_ptr_dev != nullptr){
      hipFree(slice_ptr_dev);
      CUDA_CHECK
    }

    if(step_lengths != nullptr){
      hipFree(step_lengths);
      CUDA_CHECK
    }

    if(row_starts != nullptr){
      hipFree(row_starts);
      CUDA_CHECK
    }

    if(step_starts != nullptr){
      hipFree(step_starts);
      CUDA_CHECK
    }

    if(step_blocks != nullptr){
      hipFree(step_blocks);
      CUDA_CHECK
    }

    if(mult_algo == "staircase"){
      for(int i = 0; i < num_steps; i++){
        hipStreamDestroy(streams[i]);
        CUDA_CHECK
      }
    }
}

void CUDA_SLS::smooth_sweeps(int algo, int num_sweeps, double tol, bool async){
  if(!async){
    hipFuncSetAttribute(reinterpret_cast<const void*>(smooth_kern), hipFuncAttributeMaxDynamicSharedMemorySize, smooth_smem_size);
    smooth_kern<<<blocks, tile, smooth_smem_size>>>(algo, diag_dev, entries_dev, col_dev, slice_ptr_dev, slice_mnz_dev, x_dev, b_dev, n_rows, num_sweeps, tol, x_dev_new);
    //CUDA_CHECK

    std::swap(x_dev_new, x_dev);
  }

  else{
    hipFuncSetAttribute(reinterpret_cast<const void*>(smooth_kern), hipFuncAttributeMaxDynamicSharedMemorySize, smooth_smem_size);
    smooth_kern<<<blocks, tile, smooth_smem_size>>>(algo, diag_dev, entries_dev, col_dev, slice_ptr_dev, slice_mnz_dev, x_dev, b_dev, n_rows, num_sweeps, tol);
    //CUDA_CHECK
  }

}

void CUDA_SLS::scale_iter(){
  calc_sums<<<blocks, tile>>>(diag_dev, entries_dev, col_dev, slice_ptr_dev, D_dev, R_dev, n_rows);
  //CUDA_CHECK

  scale_sell<<<blocks, tile, sizeof(double)*tile.x>>>(diag_dev, entries_dev, col_dev, slice_ptr_dev, R_dev, n_rows);
  //CUDA_CHECK
}

double CUDA_SLS::get_orig_res_sq(){
  hipMemcpy(orig_res_dev, res_dev, n_rows*sizeof(double), hipMemcpyDeviceToDevice);

  D_scale_kern<<<vector_blocks, 1024>>>(D_dev, orig_res_dev, n_rows);
  //CUDA_CHECK

  dot_prod(rTr_orig_dev, orig_res_dev);
  //CUDA_CHECK

  hipMemcpy(rTr_orig_h, rTr_orig_dev, sizeof(double), hipMemcpyDeviceToHost);
  //CUDA_CHECK

  return rTr_orig_h[0];
}

void CUDA_SLS::dot_prod(double* result, double* vec1, double* vec2){
  dot_prod_kern<<<vector_blocks, 1024>>>(vec1, n_rows, reduc_arr1, vec2);
  //CUDA_CHECK
  reduce(result);
  //CUDA_CHECK
}

void CUDA_SLS::reduce(double* result, bool max){
    //Initial length is just the number of blocks needed to reduce vector of n_rows entries
    //and this number is stored in vector_block.x
    int length = vector_blocks.x;

    //In turn, the number of blocks we need to reduce a vector of length vector_blocks.x is calculated
    int num_blocks = (length + 1023)/1024;


    if(!max){
      if(length > 1024){
          block_reduce<<<num_blocks, 1024>>>(reduc_arr1, reduc_arr2, length);
          length = num_blocks;
          block_reduce<<<1, 1024>>>(reduc_arr2, result, length);
      }

      else{
        block_reduce<<<1, 1024>>>(reduc_arr1, result, length);
      }
    }

    else{
      if(length > 1024){
          block_reduce<<<num_blocks, 1024>>>(reduc_arr1, reduc_arr2, length, true);
          length = num_blocks;
          block_reduce<<<1, 1024>>>(reduc_arr2, result, length, true);
      }

      else{
        block_reduce<<<1, 1024>>>(reduc_arr1, result, length, true);
      }
    }
}

__global__ void staircase_mult(int length, int row_start, int step_start, int n_rows){
  int read_idx = step_start + length*blockDim.x*blockIdx.x + threadIdx.x;
  int write_idx = row_start + blockIdx.x*blockDim.x + threadIdx.x;

  double row_sum = 0;

  double entry;
  int col_id;
  double x_i;
  for(int i = 0; i < length; i++){
      col_id = col_ptr[read_idx];
      entry = entries_ptr[read_idx];
      if(col_id > 0){
        x_i = x_ptr[col_id];
        row_sum += entry*x_i;
      }
      read_idx += blockDim.x;
  }

  if(write_idx < n_rows){
    y_ptr[write_idx] = row_sum;
  }
}

//Sliced ELLPACK row-vertical storage
__global__ void ellpack_shared(double* diag, double* entries, int* col_idx, int* slice_ptr, double* x_dev, double* res, int n_rows){
  extern __shared__ double x_diag[];

  int read_start = slice_ptr[blockIdx.x];
  int read_end = slice_ptr[blockIdx.x + 1];
  int num_cols = (read_end - read_start)/blockDim.x;
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int read_idx = read_start + threadIdx.x;
  bool in_domain = (row < n_rows);

  int col_id;
  double entry = (in_domain) ? diag[row] : 0;
  double x_i = (in_domain) ? x_dev[row]: 0;

  x_diag[threadIdx.x] = x_i;
  double row_sum = x_i*entry;

  __syncthreads();

  for(int i = 0; i < num_cols; i++){
    col_id = col_idx[read_idx];
    entry = entries[read_idx];

    if(col_id > 0){
      x_i = x_dev[col_id - 1];
    }

    else if(col_id < 0){
      x_i = x_diag[-(col_id + 1)];
    }

    row_sum += entry*x_i;
    read_idx += blockDim.x;

    __syncwarp();
  }

  if(in_domain){
    res[row] = row_sum;
  }
}

__global__ void max_deviation_kern(double* R, int length, double* reduc_arr){
  __shared__ double smem[1024];
  int gl_id = blockIdx.x*blockDim.x + threadIdx.x;

  if(gl_id < length){
    smem[threadIdx.x] = fabs(1 - R[gl_id]);
  }

  else{
    smem[threadIdx.x] = 0;
  }

  block_reduce_device(smem, true);

  if(threadIdx.x == 0){
    reduc_arr[blockIdx.x] = smem[0];
  }
}

//Dot product kernel (process 1024 entries in a thread-block)
__global__ void dot_prod_kern(double* vec1, int length, double* reduc_arr, double* vec2){
  __shared__ double smem[1024];
  int gl_id = blockIdx.x*blockDim.x + threadIdx.x;

  if(gl_id < length){
    double x_i = vec1[gl_id];
    double y_i = (vec2 != nullptr) ? vec2[gl_id] : x_i;
    smem[threadIdx.x] = x_i*y_i;
  }

  else{
    smem[threadIdx.x] = 0;
  }

  block_reduce_device(smem);

  if(threadIdx.x == 0){
    reduc_arr[blockIdx.x] = smem[0];
  }
}

__device__ void warpReduce_sum(volatile double* sdata){
    double vsum = sdata[threadIdx.x] + sdata[threadIdx.x+32];
    vsum += __shfl_down_sync(0xffffffff, vsum, 16);
    vsum += __shfl_down_sync(0xffffffff, vsum, 8);
    vsum += __shfl_down_sync(0xffffffff, vsum, 4);
    vsum += __shfl_down_sync(0xffffffff, vsum, 2);
    vsum += __shfl_down_sync(0xffffffff, vsum, 1);

    if (threadIdx.x == 0) {
      sdata[0] = vsum;
    }
}

__device__ void warpReduce_max(volatile double* sdata){
    double vmax = fmax(sdata[threadIdx.x], sdata[threadIdx.x+32]);
    vmax += fmax(vmax, __shfl_down_sync(0xffffffff, vmax, 16));
    vmax += fmax(vmax, __shfl_down_sync(0xffffffff, vmax, 8));
    vmax += fmax(vmax, __shfl_down_sync(0xffffffff, vmax, 4));
    vmax += fmax(vmax, __shfl_down_sync(0xffffffff, vmax, 2));
    vmax += fmax(vmax, __shfl_down_sync(0xffffffff, vmax, 1));

    if (threadIdx.x == 0) {
      sdata[0] = vmax;
    }
}

__global__ void block_reduce(double* read, double* write, int length, bool max){
    __shared__ double smem[1024];
    int gl_id = blockIdx.x*1024 + threadIdx.x;
    smem[threadIdx.x] = (gl_id < length) ? read[gl_id] : 0;

    block_reduce_device(smem, max);

    if (threadIdx.x == 0) {
      write[blockIdx.x] = smem[0];
    }
}

__device__ void block_reduce_device(volatile double* smem, bool max){
    if(max){
      __syncthreads();
      if(threadIdx.x < 512 && blockDim.x > 512){
        smem[threadIdx.x] = fmax(smem[threadIdx.x], smem[threadIdx.x + 512]);
      }
      __syncthreads();
      if(threadIdx.x < 256 && blockDim.x > 256){
        smem[threadIdx.x] = fmax(smem[threadIdx.x], smem[threadIdx.x + 256]);
      }
      __syncthreads();
      if(threadIdx.x < 128 && blockDim.x > 128){
        smem[threadIdx.x] = fmax(smem[threadIdx.x], smem[threadIdx.x + 128]);
      }
      __syncthreads();
      if(threadIdx.x < 64 && blockDim.x > 64){
        smem[threadIdx.x] = fmax(smem[threadIdx.x], smem[threadIdx.x + 64]);
      }
      __syncthreads();
      if (threadIdx.x < 32) {
        warpReduce_max(smem);
      }
      __syncthreads();
    }

    else{
      __syncthreads();
      if(threadIdx.x < 512 && blockDim.x > 512){
        smem[threadIdx.x] += smem[threadIdx.x + 512];
      }
      __syncthreads();
      if(threadIdx.x < 256 && blockDim.x > 256){
        smem[threadIdx.x] += smem[threadIdx.x + 256];
      }
      __syncthreads();
      if(threadIdx.x < 128 && blockDim.x > 128){
        smem[threadIdx.x] += smem[threadIdx.x + 128];
      }
      __syncthreads();
      if(threadIdx.x < 64 && blockDim.x > 64){
        smem[threadIdx.x] += smem[threadIdx.x + 64];
      }
      __syncthreads();
      if (threadIdx.x < 32) {
        warpReduce_sum(smem);
      }
      __syncthreads();
    }
}

//Add two vectors
__global__ void add_kern(double* result, double* vec1, double* vec2, int length, bool minus, double* scale1, double* scale2){
  double num = (scale1 != nullptr) ? scale1[0] : 1;
  double denom = (scale2 != nullptr) ? scale2[0] : 1;
  int gl_id = blockIdx.x*blockDim.x + threadIdx.x;

  double scale = (1 - 2 * minus)*num/denom;

  __syncthreads();

  if(gl_id < length){
    double vec1_i = vec1[gl_id];
    double vec2_i = scale*vec2[gl_id];
    result[gl_id] = vec1_i + vec2_i;
  }
}

__global__ void Dinv_scale_kern(double* D, double* vec, int length){
  int gl_id = blockIdx.x*1024 + threadIdx.x;
  if(gl_id < length){
    double D_i = D[gl_id];
    vec[gl_id] /= sqrt(D_i);
  }
}

__global__ void D_scale_kern(double* D, double* vec, int length){
  int gl_id = blockIdx.x*1024 + threadIdx.x;
  if(gl_id < length){
    double D_i = D[gl_id];
    vec[gl_id] *= sqrt(D_i);
  }
}

__global__ void smooth_kern(int algo, double* diag, double* entries, int* col_idx, int* slice_ptr, int* slice_mnz_arr, double* x_dev, double* b_dev, int n_rows, int max_iter, double tol, double* x_dev_new){
  extern __shared__ char smem_u[];

  int read_start = slice_ptr[blockIdx.x];
  int read_end = slice_ptr[blockIdx.x + 1];
  int mnz = slice_mnz_arr[blockIdx.x];
  int num_cols = (read_end - read_start)/blockDim.x;
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int read_idx = read_start + threadIdx.x;
  int write_idx = threadIdx.x;

  double* smem = reinterpret_cast<double*>(smem_u);
  int* loc_nbrs = reinterpret_cast<int*>(smem_u + blockDim.x * sizeof(double));
  double* loc_nbrs_coef = reinterpret_cast<double*>(smem_u + blockDim.x *(sizeof(double) + mnz * sizeof(int)));
  bool in_domain = (row < n_rows);

  double x_i = (in_domain) ? x_dev[row] : 0;
  double diag_entry = (in_domain) ? diag[row] : 1;
  double b_i = (in_domain) ? b_dev[row] : 0;

  //Find first residual square norm

  b_i -= diag_entry*x_i;

  smem[threadIdx.x] = x_i;

  //Prepare SELL arrays for local block matrix
  for(int i = 0; i < mnz; i++){
    loc_nbrs[i*blockDim.x + threadIdx.x] = -1;
    loc_nbrs_coef[i*blockDim.x + threadIdx.x] = 0;
  }

  __syncthreads();

  //Get RHS of local problem
  for(int i = 0; i < num_cols; i++){
    int col_id = col_idx[read_idx];
    double entry = entries[read_idx];

    //Check if col idx is outside block
    if(col_id > 0){
      double x_coef = x_dev[col_id-1];
      b_i -= entry*x_coef;
    }

    //If it is in local block, add index and coef
    else if (col_id != 0){
      loc_nbrs[write_idx] = -(col_id+1);
      loc_nbrs_coef[write_idx] = entry;
      b_i -= smem[-(col_id+1)]*entry;
      write_idx += blockDim.x;
    }

    read_idx += blockDim.x;
  }

  __syncthreads();

  //Find first residual square norm
  smem[threadIdx.x] = b_i*b_i;
  block_reduce_device(smem);
  double res_sq = smem[0];

  double y_i = 0;

  __syncthreads();

  if(algo == 0 && res_sq > tol){
    y_i = CG_device(smem, loc_nbrs, loc_nbrs_coef, res_sq, b_i, diag_entry, mnz, tol, max_iter);
  }

  else if(algo == 1 && res_sq > tol){
    y_i = Jacobi_device(smem, loc_nbrs, loc_nbrs_coef, b_i, diag_entry, mnz, tol, max_iter);
  }

  else if(algo == 2 && res_sq > tol){
    y_i = GS_device(smem, loc_nbrs, loc_nbrs_coef, b_i, diag_entry, mnz, tol, max_iter);
  }

  if(x_dev_new == nullptr){
    if(in_domain){
      x_dev[row] = y_i + x_i;
    }

    __threadfence_system();
  }

  else{
    if(in_domain){
      x_dev_new[row] = y_i + x_i;
    }
  }
}

__device__ double CG_device(volatile double* smem, int* loc_nbrs, double* loc_nbrs_coef, double res_sq, double d_i, double diag_entry, int mnz, double tol, int max_iter){
    int iter_count = 0;
    double res_i = d_i;
    double x_i = 0;

     while(iter_count < max_iter){
      smem[threadIdx.x] = d_i;
      __syncthreads();
      //Compute matrix-vector product
      double Ad_i = diag_entry*d_i;
      int read_idx = threadIdx.x;
      for(int i = 0; i < mnz; i++){
        int col_id = loc_nbrs[read_idx];

        if(col_id >= 0){
          Ad_i += loc_nbrs_coef[read_idx]*smem[col_id];
        }
        read_idx += blockDim.x;
      }
      __syncthreads();

      smem[threadIdx.x] = Ad_i*d_i;
      block_reduce_device(smem);

      double alpha = res_sq/smem[0];
      x_i += alpha*d_i;
      res_i -= alpha*Ad_i;

      __syncthreads();
      smem[threadIdx.x] = res_i*res_i;
      block_reduce_device(smem);
      double new_res_sq = smem[0];
      __syncthreads();

      if(new_res_sq < tol){
        return x_i;
      }

      d_i = res_i + (new_res_sq/res_sq)*d_i;
      res_sq = new_res_sq;
      iter_count += 1;
    }

    return x_i;
}

__device__ double Jacobi_device(volatile double* smem, int* loc_nbrs, double* loc_nbrs_coef, double b_i, double diag_entry, int mnz, double tol, int max_iter){
  int iter_count = 0;
  double x_i = b_i / diag_entry;

  while(iter_count < max_iter){
    smem[threadIdx.x] = x_i;
    __syncthreads();
    int read_idx = threadIdx.x;
    double res_i = b_i - x_i*diag_entry;
    for(int i = 0; i < mnz; i++){
      int col_id = loc_nbrs[read_idx];
      if(col_id >= 0){
        res_i -= loc_nbrs_coef[read_idx]*smem[col_id];
      }
      read_idx += blockDim.x;
    }

    __syncthreads();
    smem[threadIdx.x] = res_i*res_i;
    block_reduce_device(smem);
    double res_sq = smem[0];

    __syncthreads();

    if(res_sq < tol){
      return x_i;
    }

    x_i += res_i/diag_entry;
    iter_count += 1;
  }

  return x_i;
}

__device__ double GS_device(volatile double* smem, int* loc_nbrs, double* loc_nbrs_coef, double b_i, double diag_entry, int mnz, double tol, int max_iter){
  int iter_count = 0;
  double x_i = 0;
  int num_warps = blockDim.x/32;

  while(iter_count < max_iter){
    int read_idx = threadIdx.x;
    __syncthreads();
    smem[threadIdx.x] = x_i;
    __syncthreads();
    for(int i = 0; i < num_warps; i++){
      if(threadIdx.x/32 == i){
        double sum = b_i;
        for(int j = 0; j < mnz; j++){
          int col_id = loc_nbrs[read_idx];

          if(col_id >= 0){
            sum -= loc_nbrs_coef[read_idx]*smem[col_id];
          }
          read_idx += blockDim.x;
        }
        x_i = sum/diag_entry;
        smem[threadIdx.x] = x_i;
      }
      __syncthreads();
    }

    double res_i = b_i - x_i*diag_entry;
    read_idx = threadIdx.x;
    for(int j = 0; j < mnz; j++){
      int col_id = loc_nbrs[read_idx];

      if(col_id >= 0){
        res_i -= loc_nbrs_coef[read_idx]*smem[col_id];
      }
      read_idx += blockDim.x;
    }

    __syncthreads();

    smem[threadIdx.x] = res_i*res_i;
    block_reduce_device(smem);
    double res_sq = smem[0];
    __syncthreads();


    if(res_sq < tol){
      return x_i;
    }

    iter_count += 1;
  }

  return x_i;
}

__global__ void calc_sums(double* diag, double* entries, int* col_idx, int* slice_ptr, double* D, double* R, int n_rows){
  int read_start = slice_ptr[blockIdx.x];
  int read_end = slice_ptr[blockIdx.x + 1];
  int num_cols = (read_end - read_start)/blockDim.x;
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int read_idx = read_start + threadIdx.x;

  double entry = diag[row];
  double row_norm = entry*entry;

  for(int i = 0; i < num_cols; i++){
    entry = entries[read_idx];
    row_norm += entry*entry;
    read_idx += blockDim.x;
  }

  __syncthreads();

  if(row < n_rows){
    double scaler = sqrt(row_norm);
    R[row] = scaler;
    D[row] *= scaler;
  }
}

__global__ void scale_sell(double* diag, double* entries, int* col_idx, int* slice_ptr, double* R, int n_rows){
  extern __shared__ double R_local[];

  int read_start = slice_ptr[blockIdx.x];
  int read_end = slice_ptr[blockIdx.x + 1];
  int num_cols = (read_end - read_start)/blockDim.x;
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int read_idx = read_start + threadIdx.x;

  double R_i, R_row;
  int col_id;
  if(row < n_rows){
    R_i = sqrt(R[row]);
    R_row = R_i;
    R_local[threadIdx.x] = R_i;
    diag[row] /= R_i*R_i;
  }

  for(int i = 0; i < num_cols; i++){
    __syncwarp();

    col_id = col_idx[read_idx];

    if(col_id > 0){
      R_i = sqrt(R[col_id - 1]);
    }

    else if(col_id < 0){
      R_i = R_local[-(col_id + 1)];
    }

    entries[read_idx] /= R_i*R_row;
    read_idx += blockDim.x;
  }
}

__global__ void scale_inv_sell(double* diag, double* entries, int* col_idx, int* slice_ptr, double* R, int n_rows){
  extern __shared__ double R_local[];

  int read_start = slice_ptr[blockIdx.x];
  int read_end = slice_ptr[blockIdx.x + 1];
  int num_cols = (read_end - read_start)/blockDim.x;
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int read_idx = read_start + threadIdx.x;

  double R_i, R_row;
  int col_id;
  if(row < n_rows){
    R_i = sqrt(R[row]);
    R_row = R_i;
    R_local[threadIdx.x] = R_i;
    diag[row] *= R_i*R_i;
  }

  for(int i = 0; i < num_cols; i++){
    __syncwarp();

    col_id = col_idx[read_idx];

    if(col_id > 0){
      R_i = sqrt(R[col_id - 1]);
    }

    else if(col_id < 0){
      R_i = R_local[-(col_id + 1)];
    }

    entries[read_idx] *= R_i*R_row;
    read_idx += blockDim.x;
  }
}

__global__ void Chol_sell(double* A, double* A_diag, double* L, double* L_diag, int* col_idx, int* slice_ptr, double* res, int n_rows){
  int read_start = slice_ptr[blockIdx.x];
  int read_end = slice_ptr[blockIdx.x + 1];
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int read_start_i = read_start + threadIdx.x;
  int rw_idx = read_start_i;
  int num_cols = (read_end - read_start)/blockDim.x;

  //get current diagonal on this row
  double L_ii = L_diag[i];

  //iterate over non-zero elements in row i
  for (int alpha = 0; alpha < num_cols; alpha++){
    __syncthreads();

    int j = col_idx[rw_idx];

    if(j < 0){
      continue;
    }

    //Get the slice and starting index of row j
    int slice = j/blockDim.x;
    int slice_start = slice_ptr[slice];
    int slice_end = slice_ptr[slice + 1];
    int num_cols_j = (slice_end - slice_start)/blockDim.x;
    int read_start_j = slice_start + j%blockDim.x;

    //Initiate s to a_ij
    double s = A[rw_idx];

    //iterate over the non-zero elements in row j
    for(int beta = 0; beta < num_cols_j; beta++){
      int j_entry_col = col_idx[read_start_j + beta*blockDim.x];

      //check against available non-zero column indices in row i
      for (int gamma = 0; gamma < alpha; gamma++){
        int i_entry_col = col_idx[read_start_i + gamma*blockDim.x];

        if(j_entry_col == i_entry_col){
          s -= L[read_start_i + gamma*blockDim.x]*L[read_start_j + beta*blockDim.x];
        }

        else if(j_entry_col < i_entry_col){
          break;
        }
      }
    }

    L[rw_idx] = s/L_ii;
    rw_idx += blockDim.x;
  }


  L_ii = A_diag[i];

  for (int k = 0; k < num_cols; k++){
    L_ii -= pow(L[read_start_i + k*blockDim.x],2);
  }

  L_diag[i] = sqrt(L_ii);
}

__global__ void ellpack_row_based(double* entries, int* col_idx, int* slice_ptr, double* x_dev, double* res, int n_rows){
  int read_start = slice_ptr[blockIdx.x];
  int read_end = slice_ptr[blockIdx.x + 1];
  int num_cols = (read_end - read_start)/blockDim.x;
  int write_idx = blockIdx.x*blockDim.x + threadIdx.x;
  int read_idx = read_start + threadIdx.x;

  double row_sum = 0;

  for(int i = 0; i < num_cols; i++){
    int col_id = col_idx[read_idx];
    double entry = entries[read_idx];

    if(col_id >= 0){
      double x_i = x_dev[col_id];
      row_sum += entry*x_i;
    }

    read_idx += blockDim.x;
  }

  if(write_idx < n_rows){
    res[blockIdx.x*blockDim.x + threadIdx.x] = row_sum;
  }
}







